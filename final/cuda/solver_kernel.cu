#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
#include <hip/hip_vector_types.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>       // CUDA device initialization helper functions
#include "../utility/util.h"

float* d_input;
float* d_output;
float* d_dx;
float* d_dy;

texture<float, hipTextureType1D, hipReadModeElementType> tex_OriginalImage;
texture<float, hipTextureType1D, hipReadModeElementType> tex_Image;
texture<float, hipTextureType1D, hipReadModeElementType> tex_dx;
texture<float, hipTextureType1D, hipReadModeElementType> tex_dy;

__global__ void
dummy(){
}

__global__ void
test(float *input, float *output, int w, int h){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= w || y >= h){
    return;
  }
 
  int index = y * w + x;
  float texR = tex1Dfetch(tex_Image, index);
  float texlR = tex1Dfetch(tex_Image, index - 1);
  float texrR = tex1Dfetch(tex_Image, index + 1);
  //output[index] = input[index];
  output[index] = (texR + texlR + texrR) / 3;
}

__global__ void
nabla(float *dx, float *dy, int w, int h){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= w || y >= h){
    return;
  }
 
  int index = y * w + x;

  float v_dx = 0.0;
  float v_dy = 0.0;
  if(x < w - 1){
    v_dx = tex1Dfetch(tex_Image, index + 1) - tex1Dfetch(tex_Image, index);  
  }
  if(y < h - 1){
    v_dy = tex1Dfetch(tex_Image, index + w) - tex1Dfetch(tex_Image, index);  
  }

  dx[index] = v_dx;
  dy[index] = v_dy;
}

__global__ void
rof_part1(float *dx, float *dy, int w, int h, float L2, float Tau, float Sigma, float Theta){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= w || y >= h){
    return;
  }
 
  int index = y * w + x;

  float imagecopy = tex1Dfetch(tex_Image, index);
  float dx_v = tex1Dfetch(tex_dx, index);
  float dy_v = tex1Dfetch(tex_dy, index);
  float dx_inner_v = 0.0;
  float dy_inner_v = 0.0;
  float norm;

  if(x < w - 1){
    dx_inner_v = (tex1Dfetch(tex_Image, index + 1) - imagecopy) * Sigma;
  }
  if(y < h - 1){
    dy_inner_v = (tex1Dfetch(tex_Image, index + w) - imagecopy) * Sigma;
  }  
  dx_v += dx_inner_v; 
  dy_v += dy_inner_v;
  norm = max(sqrt(dx_v * dx_v + dy_v * dy_v), 1.0);
  dx_v /= norm; 
  dy_v /= norm;

  dx[index] = dx_v;
  dy[index] = dy_v;
}

__global__ void
rof_part2(float *output, int w, int h, float L2, float Tau, float Sigma, float Theta, float lt){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= w || y >= h){
    return;
  }
 
  int index = y * w + x;
  float nt = 0.0;
  float imagecopy = tex1Dfetch(tex_Image, index);
  if(x <= w - 1){
    nt -= tex1Dfetch(tex_dx, index);
  }
  if(x >= 1){
    nt += tex1Dfetch(tex_dx, index - 1);
  }
  if(y < h - 1){
    nt -= tex1Dfetch(tex_dy, index);
  }
  if(y >= 1){
    nt += tex1Dfetch(tex_dy, index - w);
  }

  float x1 = (imagecopy - nt * Tau + lt * tex1Dfetch(tex_OriginalImage, index)) / (1.0 + lt);
  output[index] = x1 + Theta * (x1 - imagecopy);
}

__global__ void
tvl1_part2(float *output, int w, int h, float L2, float Tau, float Sigma, float Theta, float shrink){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if(x >= w || y >= h){
    return;
  }
 
  int index = y * w + x;
  float nt = 0.0;
  float imagecopy = tex1Dfetch(tex_Image, index);
  if(x < w - 1){
    nt -= tex1Dfetch(tex_dx, index);
  }
  if(x >= 1){
    nt += tex1Dfetch(tex_dx, index - 1);
  }
  if(y < h - 1){
    nt -= tex1Dfetch(tex_dy, index);
  }
  if(y >= 1){
    nt += tex1Dfetch(tex_dy, index - w);
  }

  float temp = imagecopy - nt * Tau;
  float x1 = temp + max(min(tex1Dfetch(tex_OriginalImage, index) - temp, shrink), -shrink);
  output[index] = x1 + Theta * (x1 - imagecopy);
}

extern "C" void
launch_rof(float* data, float* result, int width, int height){
  //allocate resouces
  //Allocate the device input and output 
  checkCudaErrors(hipMalloc(&d_input, sizeof(float) * width * height));  
  checkCudaErrors(hipMalloc(&d_output, sizeof(float) * width * height));
  checkCudaErrors(hipMalloc(&d_dx, sizeof(float) * width * height));
  checkCudaErrors(hipMalloc(&d_dy, sizeof(float) * width * height));

  //copy memory from host to device
  checkCudaErrors(hipMemcpy(d_input, data, sizeof(float) * width * height, hipMemcpyHostToDevice));

  //bind texture
  hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
  checkCudaErrors(hipBindTexture(0, tex_OriginalImage, d_input, desc, width * height * sizeof(float)));
  checkCudaErrors(hipBindTexture(0, tex_Image, d_input, desc, width * height * sizeof(float)));

  //initialization
  {
    int threadCounts = 16;
    dim3 cudaBlockSize((width + threadCounts - 1) / threadCounts, (height + threadCounts - 1) / threadCounts, 1);
    dim3 cudaGridSize(threadCounts, threadCounts, 1);
    nabla<<<cudaBlockSize, cudaGridSize>>>(d_dx, d_dy, width, height);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipBindTexture(0, tex_dx, d_dx, desc, width * height * sizeof(float)));
    checkCudaErrors(hipBindTexture(0, tex_dy, d_dy, desc, width * height * sizeof(float)));
  }
  
  double fL2 = 8.0;
  double fTau = 0.02; 
  double fSigma = 1.0 / (fL2 * fTau);  
  double fTheta = 1.0;
  double lt = 8.0 * fTau;
  //execution
  
  for(int i = 0; i < 101; i++){
    int threadCounts = 16;
    dim3 cudaBlockSize((width + threadCounts - 1) / threadCounts, (height + threadCounts - 1) / threadCounts, 1);
    dim3 cudaGridSize(threadCounts, threadCounts, 1);
    //part1
    rof_part1<<<cudaBlockSize, cudaGridSize>>>(d_dx, d_dy, width, height, fL2, fTau, fSigma, fTheta);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipBindTexture(0, tex_dx, d_dx, desc, width * height * sizeof(float)));
    checkCudaErrors(hipBindTexture(0, tex_dy, d_dy, desc, width * height * sizeof(float)));

    //part2
    rof_part2<<<cudaBlockSize, cudaGridSize>>>(d_output, width, height, fL2, fTau, fSigma, fTheta, lt);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipBindTexture(0, tex_Image, d_output, desc, width * height * sizeof(float)));
  }

  //copy memory from device to host
  checkCudaErrors(hipMemcpy(result, d_output, sizeof(float) * width * height, hipMemcpyDeviceToHost));

  //release resources
  checkCudaErrors(hipUnbindTexture(tex_OriginalImage));
  checkCudaErrors(hipUnbindTexture(tex_Image));
  checkCudaErrors(hipUnbindTexture(tex_dx));
  checkCudaErrors(hipUnbindTexture(tex_dy));
  checkCudaErrors(hipFree(d_input));
  checkCudaErrors(hipFree(d_output));
  checkCudaErrors(hipFree(d_dx));
  checkCudaErrors(hipFree(d_dy));
}

extern "C" void
launch_tvl1(float* data, float* result, int width, int height){
  //allocate resouces
  //Allocate the device input and output 
  checkCudaErrors(hipMalloc(&d_input, sizeof(float) * width * height));  
  checkCudaErrors(hipMalloc(&d_output, sizeof(float) * width * height));
  checkCudaErrors(hipMalloc(&d_dx, sizeof(float) * width * height));
  checkCudaErrors(hipMalloc(&d_dy, sizeof(float) * width * height));

  //copy memory from host to device
  checkCudaErrors(hipMemcpy(d_input, data, sizeof(float) * width * height, hipMemcpyHostToDevice));

  //bind texture
  hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
  checkCudaErrors(hipBindTexture(0, tex_OriginalImage, d_input, desc, width * height * sizeof(float)));
  checkCudaErrors(hipBindTexture(0, tex_Image, d_input, desc, width * height * sizeof(float)));

  //initialization
  {
    int threadCounts = 16;
    dim3 cudaBlockSize((width + threadCounts - 1) / threadCounts, (height + threadCounts - 1) / threadCounts, 1);
    dim3 cudaGridSize(threadCounts, threadCounts, 1);
    nabla<<<cudaBlockSize, cudaGridSize>>>(d_dx, d_dy, width, height);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipBindTexture(0, tex_dx, d_dx, desc, width * height * sizeof(float)));
    checkCudaErrors(hipBindTexture(0, tex_dy, d_dy, desc, width * height * sizeof(float)));
  }

  double fL2 = 8.0;
  double fTau = 0.02; 
  double fSigma = 1.0 / (fL2 * fTau);  
  double fTheta = 1.0;
  double shrink = 1.0 * fTau;
  //execution
  
  for(int i = 0; i < 101; i++){
    int threadCounts = 16;
    dim3 cudaBlockSize((width + threadCounts - 1) / threadCounts, (height + threadCounts - 1) / threadCounts, 1);
    dim3 cudaGridSize(threadCounts, threadCounts, 1);
    //part1
    rof_part1<<<cudaBlockSize, cudaGridSize>>>(d_dx, d_dy, width, height, fL2, fTau, fSigma, fTheta);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipBindTexture(0, tex_dx, d_dx, desc, width * height * sizeof(float)));
    checkCudaErrors(hipBindTexture(0, tex_dy, d_dy, desc, width * height * sizeof(float)));

    //part2
    tvl1_part2<<<cudaBlockSize, cudaGridSize>>>(d_output, width, height, fL2, fTau, fSigma, fTheta, shrink);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipBindTexture(0, tex_Image, d_output, desc, width * height * sizeof(float)));
  }

  //copy memory from device to host
  checkCudaErrors(hipMemcpy(result, d_output, sizeof(float) * width * height, hipMemcpyDeviceToHost));

  //release resources
  checkCudaErrors(hipUnbindTexture(tex_OriginalImage));
  checkCudaErrors(hipUnbindTexture(tex_Image));
  checkCudaErrors(hipUnbindTexture(tex_dx));
  checkCudaErrors(hipUnbindTexture(tex_dy));
  checkCudaErrors(hipFree(d_input));
  checkCudaErrors(hipFree(d_output));
  checkCudaErrors(hipFree(d_dx));
  checkCudaErrors(hipFree(d_dy));
}

extern "C" void
launch_dummy(){
  int threadCounts = 16;
  dim3 cudaBlockSize(1, 1, 1);
  dim3 cudaGridSize(threadCounts, threadCounts, 1);
  dummy<<<cudaBlockSize, cudaGridSize>>>();
  checkCudaErrors(hipDeviceSynchronize());
}
